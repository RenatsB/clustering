#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector>
#include "gpuImageGen.h"
#include <iostream>

// for devices of compute capability 2.0 and higher
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
   #define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif

__device__ float smoothNoiseP(const thrust::device_ptr<float> d_noise,
                             const size_t width,
                             const size_t height,
                             const float x,
                             const float y)
{
    //get fractional part of x and y
    float fractX = x - int(x);
    float fractY = y - int(y);

    //wrap around
    int x1 = (int(x) + width) % width;
    int y1 = (int(y) + height) % height;

    //neighbor values
    int x2 = (x1 + width - 1) % width;
    int y2 = (y1 + height - 1) % height;

    //smooth the noise with bilinear interpolation
    float value;
    value  = fractX       * fractY       * d_noise[y1*width+x1];
    value += (1 - fractX) * fractY       * d_noise[y1*width+x2];
    value += fractX       * (1 - fractY) * d_noise[y2*width+x1];
    value += (1 - fractX) * (1 - fractY) * d_noise[y2*width+x2];

    return value;
}

__device__ float turbulenceP(const thrust::device_ptr<float> d_noise,
                             const size_t imageWidth,
                             const size_t imageHeight,
                             const float x,
                             const float y,
                             const float size)
{
    float value = 0.0, localSize = size;

    while(localSize >= 1)
    {
      value += smoothNoiseP(d_noise, imageWidth, imageHeight, x / localSize, y / localSize) * localSize;
      localSize /= 2.0;
    }

    return(128.0 * value / size)/256.0;
    //return(value / size);
}

__global__ void assignColorsP(thrust::device_ptr<float> d_noise,
                              const size_t data_size,
                              const size_t imageWidth,
                              const size_t imageHeight,
                              const float turbulence_size,
                              thrust::device_ptr<float> d_out,
                              const bool randAlpha)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= data_size) return;

    const size_t x = index%imageWidth;
    const size_t y = (index-x)/imageWidth;

    d_out[index*4]   = turbulenceP(d_noise, imageWidth, imageHeight, x, y, turbulence_size);
    d_out[index*4+1] = turbulenceP(d_noise, imageWidth, imageHeight, x, y+imageHeight, turbulence_size/2);
    d_out[index*4+2] = turbulenceP(d_noise, imageWidth, imageHeight, x, y+imageHeight*2, turbulence_size/2);
    d_out[index*4+3] = 1.0f;
    if(randAlpha)
        d_out[index*4+3] = turbulenceP(d_noise, imageWidth, imageHeight, imageWidth-x, y, turbulence_size);
}

__global__ void assignColors4(thrust::device_ptr<float> d_noise,
                              const size_t data_size,
                              const size_t imageWidth,
                              const size_t imageHeight,
                              const float turbulence_size,
                              thrust::device_ptr<float> d_outR,
                              thrust::device_ptr<float> d_outG,
                              thrust::device_ptr<float> d_outB,
                              thrust::device_ptr<float> d_outA,
                              const bool randAlpha)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= data_size) return;

    const size_t x = index%imageWidth;
    const size_t y = (index-x)/imageWidth;

    d_outR[index] = turbulenceP(d_noise, imageWidth, imageHeight, x, y, turbulence_size);
    d_outG[index] = turbulenceP(d_noise, imageWidth, imageHeight, x, y+imageHeight, turbulence_size/2);
    d_outB[index] = turbulenceP(d_noise, imageWidth, imageHeight, x, y+imageHeight*2, turbulence_size/2);
    d_outA[index] = 1.0f;
    if(randAlpha)
        d_outA[index] = turbulenceP(d_noise, imageWidth, imageHeight, imageWidth-x, y, turbulence_size);
}

ColorVector gpuImageGen::generate_parallel_CV(const size_t w,
                                              const size_t h,
                                              const size_t turbulence_size,
                                              const size_t numThreads,
                                              const bool randAlpha)
{
    int dataSize = w*h;
    ColorVector outData(dataSize);
    thrust::host_vector<float> h_transfer(dataSize*4);
    thrust::device_vector<float> d_noise(dataSize);
    thrust::device_vector<float> d_colors(dataSize*4);

    const int blocks = (dataSize + numThreads - 1) / numThreads;
    //generate the per-pixel noise
    float *ptr = thrust::raw_pointer_cast(d_noise.data());
    gpuRandFn::randFloatsInternal(ptr,dataSize,numThreads);

    hipDeviceSynchronize();
    //generate the map here
    assignColorsP<<<blocks, numThreads>>>(d_noise.data(),
                                          dataSize,
                                          w,
                                          h,
                                          turbulence_size,
                                          d_colors.data(),
                                          randAlpha);
    //end of map generation
    hipDeviceSynchronize();
    thrust::copy(d_colors.begin(), d_colors.end(), h_transfer.begin());
    for(size_t i=0; i<dataSize; ++i)
    {
        outData.at(i).m_r = h_transfer[i*4];
        outData.at(i).m_g = h_transfer[i*4+1];
        outData.at(i).m_b = h_transfer[i*4+2];
        outData.at(i).m_a = h_transfer[i*4+3];
    }
    return outData;
}

ImageColors gpuImageGen::generate_parallel_IC(const size_t w,
                                              const size_t h,
                                              const size_t turbulence_size,
                                              const size_t numThreads,
                                              const bool randAlpha)
{
    int dataSize = w*h;
    ImageColors outData;
    outData.resize(dataSize);
    thrust::device_vector<float> d_noise(dataSize);
    thrust::device_vector<float> d_colorsR(dataSize);
    thrust::device_vector<float> d_colorsG(dataSize);
    thrust::device_vector<float> d_colorsB(dataSize);
    thrust::device_vector<float> d_colorsA(dataSize);

    const int blocks = (dataSize + numThreads - 1) / numThreads;

    float *ptr = thrust::raw_pointer_cast(d_noise.data());
    gpuRandFn::randFloatsInternal(ptr,dataSize,numThreads);

    hipDeviceSynchronize();
    //generate the map here
    assignColors4<<<blocks, numThreads>>>(d_noise.data(),
                                          dataSize,
                                          w,
                                          h,
                                          turbulence_size,
                                          d_colorsR.data(),
                                          d_colorsG.data(),
                                          d_colorsB.data(),
                                          d_colorsA.data(),
                                          randAlpha);
    //end of map generation
    hipDeviceSynchronize();
    thrust::copy(d_colorsR.begin(), d_colorsR.end(), outData.m_r.begin());
    thrust::copy(d_colorsG.begin(), d_colorsG.end(), outData.m_g.begin());
    thrust::copy(d_colorsB.begin(), d_colorsB.end(), outData.m_b.begin());
    thrust::copy(d_colorsA.begin(), d_colorsA.end(), outData.m_a.begin());
    return outData;
}

std::vector<float> gpuImageGen::generate_parallel_LN(const size_t w,
                                                     const size_t h,
                                                     const size_t turbulence_size,
                                                     const size_t numThreads,
                                                     const bool randAlpha)
{
    int dataSize = w*h;
    std::vector<float> outData(dataSize*4);
    thrust::device_vector<float> d_noise(dataSize);
    thrust::device_vector<float> d_colors(dataSize*4);

    const int blocks = (dataSize + numThreads - 1) / numThreads;
    //generate the per-pixel noise

    float *ptr = thrust::raw_pointer_cast(d_noise.data());
    gpuRandFn::randFloatsInternal(ptr,dataSize,numThreads);

    hipDeviceSynchronize();
    //generate the map here
    assignColorsP<<<blocks, numThreads>>>(d_noise.data(),
                                          dataSize,
                                          w,
                                          h,
                                          turbulence_size,
                                          d_colors.data(),
                                          randAlpha);
    //end of map generation
    hipDeviceSynchronize();
    thrust::copy(d_colors.begin(), d_colors.end(), outData.begin());
    return outData;
}

void gpuImageGen::generate_parallel_4SV(std::vector<float>* redChannel,
                                        std::vector<float>* greenChannel,
                                        std::vector<float>* blueChannel,
                                        std::vector<float>* alphaChannel,
                                        const size_t w,
                                        const size_t h,
                                        const size_t turbulence_size,
                                        const size_t numThreads,
                                        const bool randAlpha)
{
    int dataSize = w*h;
    thrust::device_vector<float> d_noise(dataSize);
    thrust::device_vector<float> d_colRed(dataSize);
    thrust::device_vector<float> d_colGrn(dataSize);
    thrust::device_vector<float> d_colBlu(dataSize);
    thrust::device_vector<float> d_colAlp(dataSize);

    const int blocks = (dataSize + numThreads - 1) / numThreads;
    //generate the per-pixel noise

    float *ptr = thrust::raw_pointer_cast(d_noise.data());
    gpuRandFn::randFloatsInternal(ptr,dataSize,numThreads);

    hipDeviceSynchronize();
    //generate the map here
    assignColors4<<<blocks, numThreads>>>(d_noise.data(),
                                          dataSize,
                                          w,
                                          h,
                                          turbulence_size,
                                          d_colRed.data(),
                                          d_colGrn.data(),
                                          d_colBlu.data(),
                                          d_colAlp.data(),
                                          randAlpha);
    //end of map generation
    hipDeviceSynchronize();
    thrust::copy(d_colRed.begin(), d_colRed.end(), redChannel->begin());
    thrust::copy(d_colGrn.begin(), d_colGrn.end(), greenChannel->begin());
    thrust::copy(d_colBlu.begin(), d_colBlu.end(), blueChannel->begin());
    thrust::copy(d_colAlp.begin(), d_colAlp.end(), alphaChannel->begin());
    return;
}

void gpuImageGen::generate_parallel_4LV(float* redChannel,
                                        float* greenChannel,
                                        float* blueChannel,
                                        float* alphaChannel,
                                        const size_t w,
                                        const size_t h,
                                        const size_t turbulence_size,
                                        const size_t numThreads,
                                        const bool randAlpha)
{
    int dataSize = w*h;
    thrust::device_vector<float> d_noise(dataSize);
    thrust::device_vector<float> d_colRed(dataSize);
    thrust::device_vector<float> d_colGrn(dataSize);
    thrust::device_vector<float> d_colBlu(dataSize);
    thrust::device_vector<float> d_colAlp(dataSize);

    const int blocks = (dataSize + numThreads - 1) / numThreads;
    //generate the per-pixel noise

    float *ptr = thrust::raw_pointer_cast(d_noise.data());
    gpuRandFn::randFloatsInternal(ptr,dataSize,numThreads);

    hipDeviceSynchronize();
    //generate the map here
    assignColors4<<<blocks, numThreads>>>(d_noise.data(),
                                          dataSize,
                                          w,
                                          h,
                                          turbulence_size,
                                          d_colRed.data(),
                                          d_colGrn.data(),
                                          d_colBlu.data(),
                                          d_colAlp.data(),
                                          randAlpha);
    //end of map generation
    hipDeviceSynchronize();
    thrust::copy(d_colRed.begin(), d_colRed.end(), redChannel);
    thrust::copy(d_colGrn.begin(), d_colGrn.end(), greenChannel);
    thrust::copy(d_colBlu.begin(), d_colBlu.end(), blueChannel);
    thrust::copy(d_colAlp.begin(), d_colAlp.end(), alphaChannel);
    return;
}
