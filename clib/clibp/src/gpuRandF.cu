#include "gpuRandF.h"
// Cuda includes begin
#include <hip/hip_runtime.h>
#include <hiprand.h>
// cuda includes end

#define CURAND_CALL(x) {\
if((x)!=HIPRAND_STATUS_SUCCESS) {\
printf("CURAND failure at %s:%d\n",__FILE__,__LINE__);\
exit(0);\
}\
}


int GPUclib::randFloatsInternal(float *&devData, const size_t n)
{
  // The generator, used for random numbers
  hiprandGenerator_t gen;

  // Create pseudo-random number generator
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

  // Set seed to be the current time (note that calls close together will have same seed!)
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

    // Generate n floats on device
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));
  // Cleanup
  CURAND_CALL(hiprandDestroyGenerator(gen));
  return EXIT_SUCCESS;
}
